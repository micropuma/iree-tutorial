#include <iostream>
#include <cassert>
#include <cmath> // 用于 fabs
#include <hip/hip_runtime.h>

#define N 500000 // Tuned such that kernel takes a few microseconds

// CUDA kernel function
__global__ void shortKernel(float *out_d, float *in_d) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        out_d[idx] = 1.23 * in_d[idx]; // Example operation
    }
}

int main() {
    // Allocate host memory
    float *h_in = new float[N];
    float *h_out = new float[N];

    // Initialize input data on host
    for (int i = 0; i < N; ++i) {
        h_in[i] = static_cast<float>(i); // Example input: sequential numbers
    }

    // Allocate device memory
    float *d_in, *d_out;
    hipError_t err;

    err = hipMalloc((void**)&d_in, N * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error for d_in: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    err = hipMalloc((void**)&d_out, N * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error for d_out: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // Copy input data from host to device
    err = hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start);

    // Launch kernel with 256 threads per block and enough blocks to cover N elements
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock; // Ceil division
    shortKernel<<<blocksPerGrid, threadsPerBlock>>>(d_out, d_in);

    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // Synchronize to ensure kernel execution is complete
    hipDeviceSynchronize();

    // Record stop event
    hipEventRecord(stop);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << "Kernel execution time: " << elapsedTime << " ms" << std::endl;

    // Copy result back to host
    err = hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // Verify the output with a tolerance
    for (int i = 0; i < N; ++i) {
        std::cerr << h_out[i]  << std::endl;
    }

    // Clean up
    delete[] h_in;
    delete[] h_out;
    hipFree(d_in);
    hipFree(d_out);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

